// Получение информации о некоторых свойствах устройств cuda
#include <hip/hip_runtime.h>
#include <stdio.h>
int main(int argc, char *argv[])
{
    printf("Starostenkov A.A. VM-22 (mag.)\n");
    printf("Example 1.\n");

    int deviceCount;
    hipDeviceProp_t devProp;
    hipGetDeviceCount(&deviceCount);
    printf("Found %d devices\n", deviceCount);
    for (int device = 0; device < deviceCount; device++)
    {
        hipGetDeviceProperties(&devProp, device);
        printf("Device %d\n", device);
        printf("Compute capability     : %d.%d\n", devProp.major, devProp.minor);
        printf("Name                   : %s\n", devProp.name);
        printf("Total Global Memory    : %d\n", devProp.totalGlobalMem);
        printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
        printf("Registers per block    : %d\n", devProp.regsPerBlock);
        printf("Warp size              : %d\n", devProp.warpSize);
        printf("Max threads per block  : %d\n", devProp.maxThreadsPerBlock);
        printf("Max threads dimensions: x = %d, y = %d, z = %d\n",
               devProp.maxThreadsDim[0],
               devProp.maxThreadsDim[1],
               devProp.maxThreadsDim[2]);
        printf("Max grid size: x = %d, y = %d, z = %d\n",
               devProp.maxGridSize[0],
               devProp.maxGridSize[1],
               devProp.maxGridSize[2]);
        printf("Clock rate: %d\n", devProp.clockRate);
        printf("Multiprocessor count: %d\n", devProp.multiProcessorCount);
        printf("Total constant memory  : %d\n", devProp.totalConstMem);
    }

    return 0;
}
