// Поэлементное сложение векторов (несколько блоков)
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    // вычисление индекса массивов, используем несколько
    // блоков по координате x (blockDim.x),
    // в каждом блоке – несколько нитей по координате x
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

#define MIN_N (4096 * 4096 / 256) // Минимальный размер векторов
#define MAX_N (4096 * 4096 * 16)  // Максимальный размер векторов
#define MULTIPLY_N 2              // множитель
#define THREADS_PER_BLOCK 1024    // максимальное значение для GK110

int main(void)
{
    printf("Starostenkov A.A. VM-22 (mag.)\n");
    printf("Example 5.\n");

    hipEvent_t start, stop;
    float gpuTime;
    size_t n; // представления размера векторов

    printf("\n-----START-----\n");

    FILE *file = fopen("5_output.txt", "w");
    if (file == NULL)
    {
        printf("Failed to open the file.\n");
        return 1;
    }

    fprintf(file, "n: gpuTime\n");


    for (n = MIN_N; n <= MAX_N; n *= MULTIPLY_N)
    {
        printf("n = %d\n", n);

        int *a, *b, *c;             // host копии a, b, c
        int *dev_a, *dev_b, *dev_c; // device копии of a, b, c
        // int size = N * sizeof(int);
        size_t size = n * sizeof(int);

        hipEventCreate(&start);
        hipEventCreate(&stop);

        // выделяем память на device для of a, b, c
        hipMalloc((void **)&dev_a, size);
        hipMalloc((void **)&dev_b, size);
        hipMalloc((void **)&dev_c, size);
        // выделяем память на хосте
        a = (int *)malloc(size);
        b = (int *)malloc(size);
        c = (int *)malloc(size);
        // инициализация массивов
        size_t i;
        for (i = 0; i < n; ++i)
            a[i] = 10;
        for (i = 0; i < n; ++i)
            b[i] = 9;
        
        hipEventRecord(start, 0); // отсечка
        // копируем ввод на device
        hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

        // запускаем на выполнение add() kernel с блоками и нитями
        add<<<n / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c);
        
        // копируем результат работы device на host ( копия c )
        hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
        hipEventRecord(stop, 0); // отсечка
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);

        printf("c = [");
        for (i = 0; i < 20; ++i) printf("%d; ", c[i]);
        printf("]\n");

        printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

        fprintf(file, "%zu: %.2f\n", n, gpuTime);

        hipEventDestroy(start);
        hipEventDestroy(stop);

        free(a);
        free(b);
        free(c);
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
    }

    fclose(file);

    return 0;
}
