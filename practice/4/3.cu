#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) // Функция - ядро
{
    *c = *a + *b;
}

int main(void)
{
    printf("Starostenkov A.A. VM-22 (mag.)\n");
    printf("Example 3.\n");

    int a, b, c;                // host копии a, b, c
    int *dev_a, *dev_b, *dev_c; // device копии a, b, c
    int size = sizeof(int);
    // выделяем память для device копий для a, b, c
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);
    a = 10;
    b = 9;
    // копируем ввод на device
    hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);
    // запускаем add() kernel на GPU, передавая параметры
    add<<<1, 1>>>(dev_a, dev_b, dev_c);
    // копируем результат с на CPU
    hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
    // освобождаем память device
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    printf("c = %5d\n", c);
    return 0;
}
