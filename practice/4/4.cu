// Поэлементное сложение векторов (один блок)
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 128

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x; // Связываем элемент массива с глобальным номером нити
    if (tid > N - 1)
        return;
    c[tid] = a[tid] + b[tid]; // каждый tid – одна нить
}

int main()
{
    printf("Starostenkov A.A. VM-22 (mag.)\n");
    printf("Example 4.\n");

    int host_a[N], host_b[N], host_c[N];
    int *dev_a, *dev_b, *dev_c;
    for (int i = 0; i < N; i++)
    {
        host_a[i] = 10;
        host_b[i] = 9;
    }
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));
    hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice);
    add<<<1, N>>>(dev_a, dev_b, dev_c); // один блок, N потоков
    hipMemcpy(host_c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", host_a[i], host_b[i], host_c[i]);
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
