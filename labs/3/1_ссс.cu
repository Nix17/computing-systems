#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void computeSum(double* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int i = idx + 1;
        double denominator = 5 * i * i - 8.0 * i + 6.0;
        result[idx] = 5.0 / denominator;
    }
}

int main() {
    int n = 1000;  // Количество элементов в ряде
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Выделение памяти на хосте (CPU)
    double* h_result = (double*)malloc(n * sizeof(double));

    // Выделение памяти на устройстве (GPU)
    double* d_result;
    hipMalloc((void**)&d_result, n * sizeof(double));

    // Вычисление суммы на GPU
    computeSum<<<gridSize, blockSize>>>(d_result, n);

    // Копирование результата с GPU на CPU
    hipMemcpy(h_result, d_result, n * sizeof(double), hipMemcpyDeviceToHost);

    // Вывод результата
    double sum = 0.0;
    for (int i = 0; i < n; i++) {
        sum += h_result[i];
    }
    printf("Sum: %f\n", sum);

    // Освобождение памяти
    free(h_result);
    hipFree(d_result);

    return 0;
}
