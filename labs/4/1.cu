#include <hip/hip_runtime.h>
#include <stdio.h>

// Переопределяем количество итераций цикла
// #define N 1000000

// another
// #define N 100
// #define N 1000
// #define N 10000
// #define N 100000
// #define N 1000000
#define N 10000000
// Переопределяем максимальное значение для GK110
#define THREADS_PER_BLOCK 1024

// Функция - ядро
__global__ void get_el(float *dev_el)
{
    float index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index <= N)
    {
        float znam = (10 * (index + 1) * (index + 1) - 2 * (index + 1) - 3);
        dev_el[(int)index] = 3.0 / znam;
    }
    else
    {
        return;
    }
}

int main(void)
{
    float gpu_calc_time, gpu_send_time;
    hipEvent_t start, stop;

    // Создание событий
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host копии
    float *el;
    // Device копии
    float *dev_el;

    // Выделение памяти для device и host элементов
    int size = N * sizeof(float);
    el = (float *)malloc(size);
    hipMalloc((void **)&dev_el, size);

    // Отсечка
    hipEventRecord(start, 0);
    get_el<<<(int)(N / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(dev_el);

    // Отсечка
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_calc_time, start, stop);

    // Копирование результата в CPU
    // Отсечка
    hipEventRecord(start, 0);
    hipMemcpy(el, dev_el, size, hipMemcpyDeviceToHost);
    // Отсечка
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_send_time, start, stop);

    // Освобождение памяти device
    hipFree(dev_el);
    // Переменные времени и суммы
    float cpu_t;
    double sum = 0;

    // Вычисление суммы числового ряда
    // Отсечка
    hipEventRecord(start, 0);
    for (float i = 0; i < N; i++)
    {
        sum += el[(int)i];
    }
    // Отсечка
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_t, start, stop);

    printf("Количество итераций: %d\n", N);
    printf("Cумма числового ряда: %.6f\n", sum);
    printf("Время вычисления в gpu: %.4f мс\n", gpu_calc_time);
    printf("Время пересылок: %.4f мс\n", gpu_send_time);
    printf("Время суммирования в cpu: %.4f мс\n", cpu_t);
    printf("Общее время: %.4f мс\n", gpu_calc_time + gpu_send_time + cpu_t);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
